
#include "kernel_wrappers/common.h"
#include "util/endianness.h"
#ifdef __HIPCC__
#include "kernels/decompression/discard_zero_bytes_fixed.cuh"
#endif

namespace cuda {
namespace kernels {
namespace decompression {
namespace discard_zero_bytes {
namespace fixed_width {

using util::endianness_t;

#ifndef __HIPCC__
enum : serialization_factor_t { DefaultSerializationFactor = 32 };
#endif

// TODO: This currently ignores the possibility of a sorted variant of the kernel

template<
	unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize,
	endianness_t UncompressedEndianness  = util::compilation_target_endianness(),
	serialization_factor_t SerializationFactor = DefaultSerializationFactor>
class kernel_t : public cuda::registered::kernel_t {
public:
	REGISTERED_KERNEL_WRAPPER_BOILERPLATE_DEFINITIONS(kernel_t);

	using uncompressed_type = util::uint_t<UncompressedSize>;
	using compressed_type = util::uint_t<CompressedSize>;

	launch_configuration_t resolve_launch_configuration(
		device::properties_t              device_properties,
		device_function::attributes_t     kernel_function_attributes,
		size_t                            length,
		launch_configuration_limits_t     limits) const
#ifdef __HIPCC__
	{
		launch_config_resolution_params_t<
			IndexSize, UncompressedSize, CompressedSize,
			translate(UncompressedEndianness), SerializationFactor
		> params(
			device_properties,
			length);

		return cuda::kernels::resolve_launch_configuration(params, limits);
	}
#else
	;
#endif
};

#ifdef __HIPCC__

template<
	unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize,
	endianness_t UncompressedEndianness, serialization_factor_t SerializationFactor>
launch_configuration_t kernel_t<
	IndexSize, UncompressedSize, CompressedSize, UncompressedEndianness, SerializationFactor
	>::resolve_launch_configuration(
	device::properties_t           device_properties,
	device_function::attributes_t  kernel_function_attributes,
	arguments_type                 extra_arguments,
	launch_configuration_limits_t  limits) const
{
	auto length = any_cast<size_t>(extra_arguments.at("length"));

	return resolve_launch_configuration(
		device_properties, kernel_function_attributes,
		length,
		limits);
}


template<
	unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize,
	endianness_t UncompressedEndianness, serialization_factor_t SerializationFactor>
void kernel_t<IndexSize, UncompressedSize, CompressedSize, UncompressedEndianness, SerializationFactor>::enqueue_launch(
	stream::id_t                   stream,
	const launch_configuration_t&  launch_config,
	arguments_type                 arguments) const
{
	using index_type        = util::uint_t<IndexSize>;
	using uncompressed_type = util::uint_t<UncompressedSize>;
	using compressed_type   = util::uint_t<CompressedSize>;

	auto decompressed     = any_cast<uncompressed_type*     >(arguments.at("decompressed"     ));
	auto compressed_input = any_cast<const compressed_type* >(arguments.at("compressed_input" ));
	auto length           = any_cast<index_type             >(arguments.at("length"           ));

	cuda::kernel::enqueue_launch(
		*this, stream, launch_config,
		decompressed, compressed_input, length
	);
}

template<
	unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize,
	endianness_t UncompressedEndianness, serialization_factor_t SerializationFactor>
const device_function_t kernel_t<IndexSize, UncompressedSize, CompressedSize, UncompressedEndianness, SerializationFactor>::get_device_function() const
{
	return reinterpret_cast<const void*>(cuda::kernels::decompression::discard_zero_bytes::fixed_width::decompress
		<IndexSize, UncompressedSize, CompressedSize, translate(UncompressedEndianness), SerializationFactor>);
}

static_block {
	namespace functors = cuda::functors;

	//         IndexSize  UncompressedSize  CompressedSize
	//------------------------------------------------------------------------------------
	kernel_t < 4,         2,                1 >::registerInSubclassFactory();
	kernel_t < 4,         4,                1 >::registerInSubclassFactory();
	kernel_t < 4,         4,                2 >::registerInSubclassFactory();
	kernel_t < 4,         8,                1 >::registerInSubclassFactory();
	kernel_t < 4,         8,                2 >::registerInSubclassFactory();
	kernel_t < 4,         8,                4 >::registerInSubclassFactory();

	kernel_t < 8,         2,                1 >::registerInSubclassFactory();
	kernel_t < 8,         4,                1 >::registerInSubclassFactory();
	kernel_t < 8,         4,                2 >::registerInSubclassFactory();
	kernel_t < 8,         8,                1 >::registerInSubclassFactory();
	kernel_t < 8,         8,                2 >::registerInSubclassFactory();
	kernel_t < 8,         8,                4 >::registerInSubclassFactory();
}
#endif /* __HIPCC__ */


} // namespace fixed_width
} // namespace discard_zero_bytes
} // namespace decompression
} // namespace kernels
} // namespace cuda

