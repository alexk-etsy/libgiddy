
#include "kernel_wrappers/common.h"
#include "util/endianness.h"
#ifdef __HIPCC__
#include "kernels/decompression/discard_zero_bytes_variable.cuh"
#endif

namespace cuda {
namespace kernels {
namespace decompression {
namespace discard_zero_bytes {
namespace variable_width {

using util::endianness_t;


// TODO: This currently ignores the possibility of a sorted variant of the kernel

template<unsigned IndexSize, /* util::terminal_t EndToPad, */unsigned UncompressedSize, unsigned ElementSizesContainerSize>
class kernel_t : public cuda::registered::kernel_t {
public:
	REGISTERED_KERNEL_WRAPPER_BOILERPLATE_DEFINITIONS(kernel_t);

	using element_size_t     = unsigned; // code duplication with the kernel file!
	using uncompressed_type = util::uint_t<UncompressedSize>;
	using element_sizes_container_size_type = util::uint_t<ElementSizesContainerSize>;

	launch_configuration_t resolve_launch_configuration(
		device::properties_t           device_properties,
		device_function::attributes_t  kernel_function_attributes,
		size_t                         length_in_elements,
		size_t                         position_anchoring_period,
		launch_configuration_limits_t  limits) const
#ifdef __HIPCC__
	{
		launch_config_resolution_params_t<
			IndexSize, UncompressedSize, ElementSizesContainerSize
		> params(
			device_properties,
			length_in_elements, position_anchoring_period);

		return cuda::kernels::resolve_launch_configuration(params, limits);
	}
#else
	;
#endif

};

#ifdef __HIPCC__

template<unsigned IndexSize, /* util::terminal_t EndToPad, */unsigned UncompressedSize, unsigned ElementSizesContainerSize>
launch_configuration_t kernel_t<IndexSize, UncompressedSize, ElementSizesContainerSize>::resolve_launch_configuration(
	device::properties_t            device_properties,
	device_function::attributes_t kernel_function_attributes,
	arguments_type                 extra_arguments,
	launch_configuration_limits_t  limits) const
{
	auto length_in_elements        = any_cast<size_t>(extra_arguments.at("length_in_elements"       ));
	auto position_anchoring_period = any_cast<size_t>(extra_arguments.at("position_anchoring_period"));

	return resolve_launch_configuration(
		device_properties, kernel_function_attributes,
		length_in_elements, position_anchoring_period,
		limits);
}

template<unsigned IndexSize, /* util::terminal_t EndToPad, */unsigned UncompressedSize, unsigned ElementSizesContainerSize>
void kernel_t<IndexSize, UncompressedSize, ElementSizesContainerSize>::enqueue_launch(
	stream::id_t                   stream,
	const launch_configuration_t&  launch_config,
	arguments_type                 arguments) const
{
	using index_type = uint_t<IndexSize>;
	using uncompressed_type = util::uint_t<UncompressedSize>;
	using element_sizes_container_size_type = util::uint_t<ElementSizesContainerSize>;

	auto decompressed                   = any_cast<uncompressed_type*   >(arguments.at("decompressed"                 ));
	auto compressed_data                = any_cast<const unsigned char* >(arguments.at("compressed_data"              ));
	auto packed_element_sizes           = any_cast<const element_sizes_container_size_type *
	                                                                    >(arguments.at("packed_element_sizes"         ));
	auto position_anchors               = any_cast<const index_type*    >(arguments.at("position_anchors"             ));
	auto position_anchoring_period      = any_cast<index_type           >(arguments.at("position_anchoring_period"    ));
	auto num_elements                   = any_cast<index_type           >(arguments.at("num_elements"                 ));
	auto min_represented_element_size   = any_cast<element_size_t       >(arguments.at("min_represented_element_size" ));
	auto bits_per_element_size          = any_cast<unsigned             >(arguments.at("bits_per_element_size"        ));
	cuda::kernel::enqueue_launch(
		*this, stream, launch_config,
		decompressed, compressed_data, packed_element_sizes, position_anchors,
		position_anchoring_period, num_elements, min_represented_element_size,
		bits_per_element_size
	);
}

template<unsigned IndexSize, /* util::terminal_t EndToPad, */unsigned UncompressedSize, unsigned ElementSizesContainerSize>
const device_function_t kernel_t<IndexSize, UncompressedSize, ElementSizesContainerSize>::get_device_function() const
{
	return reinterpret_cast<const void*>(cuda::kernels::decompression::discard_zero_bytes::variable_width::decompress
		<IndexSize, UncompressedSize, ElementSizesContainerSize>);
}


static_block {
	//         IndexSize  Uncompressed   ElementSizesContainerSize
	//--------------------------------------------------------------------------------------
	kernel_t < 4,         2,             4 >::registerInSubclassFactory();
	kernel_t < 4,         4,             4 >::registerInSubclassFactory();
	kernel_t < 4,         8,             4 >::registerInSubclassFactory();

	kernel_t < 8,         2,             4 >::registerInSubclassFactory();
	kernel_t < 8,         4,             4 >::registerInSubclassFactory();
	kernel_t < 8,         8,             4 >::registerInSubclassFactory();

	kernel_t < 4,         2,             8 >::registerInSubclassFactory();
	kernel_t < 4,         4,             8 >::registerInSubclassFactory();
	kernel_t < 4,         8,             8 >::registerInSubclassFactory();
}

#endif /* __HIPCC__ */

} // namespace variable_width
} // namespace discard_zero_bytes
} // namespace decompression
} // namespace kernels
} // namespace cuda
