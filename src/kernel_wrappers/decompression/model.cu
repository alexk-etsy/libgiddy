
#include "kernel_wrappers/common.h"
#include "cuda/model_functors.hpp"
#ifdef __HIPCC__
#include "kernels/decompression/model.cuh"
#endif

namespace cuda {
namespace kernels {
namespace decompression {
namespace model {

template<unsigned IndexSize, typename Uncompressed, typename UnaryModelFunction>
class kernel: public cuda::registered::kernel_t {
public:
	using model_coefficients = typename UnaryModelFunction::model_coefficients_type;
	REGISTERED_KERNEL_WRAPPER_BOILERPLATE_DEFINITIONS(kernel);

};

#ifdef __HIPCC__

template<unsigned IndexSize, typename Uncompressed, typename UnaryModelFunction>
launch_configuration_t kernel<IndexSize, Uncompressed, UnaryModelFunction>::resolve_launch_configuration(
	device::properties_t            device_properties,
	device_function::attributes_t   kernel_function_attributes,
	arguments_type                  extra_arguments,
	launch_configuration_limits_t   limits) const
{
	namespace kernel_ns = cuda::kernels::decompression::model;

	auto length = any_cast<size_t>(extra_arguments.at("length"));
	kernel_ns::launch_config_resolution_params_t<IndexSize, Uncompressed, UnaryModelFunction> params(
		device_properties, length);

	return cuda::kernels::resolve_launch_configuration(params, limits);
}


template<unsigned IndexSize, typename Uncompressed, typename UnaryModelFunction>
void kernel<IndexSize, Uncompressed, UnaryModelFunction>::enqueue_launch(
	stream::id_t                   stream,
	const launch_configuration_t&  launch_config,
	arguments_type                 arguments) const
{
	using index_type = uint_t<IndexSize>;
	using model_coefficients_type =
		typename UnaryModelFunction::model_coefficients_type;
	auto length          = any_cast<index_type>(arguments.at("length"));

	auto decompressed       = any_cast<Uncompressed*           >(arguments.at("decompressed"       ));
	auto model_coefficients = any_cast<model_coefficients_type >(arguments.at("model_coefficients" ));

	cuda::kernel::enqueue_launch(
		*this, stream, launch_config,
		decompressed, model_coefficients, length);
}

template<unsigned IndexSize, typename Uncompressed, typename UnaryModelFunction>
const device_function_t kernel<IndexSize, Uncompressed, UnaryModelFunction>::get_device_function() const
{
	return reinterpret_cast<const void*>(cuda::kernels::decompression::model::decompress
		<IndexSize, Uncompressed, UnaryModelFunction>);
}


static_block {
	namespace functors = cuda::functors;
	namespace unary_models = ::cuda::functors::unary::parametric_model;

	//       IndexSize  Uncompressed   UnaryModelFunction
	//-------------------------------------------------------------------------------
	kernel < 4,         int16_t,       unary_models::linear  < 4, int16_t >  >::registerInSubclassFactory();
	kernel < 4,         int32_t,       unary_models::linear  < 4, int32_t >  >::registerInSubclassFactory();
	kernel < 8,         int32_t,       unary_models::linear  < 8, int32_t >  >::registerInSubclassFactory();
	kernel < 4,         int16_t,       unary_models::constant< 4, int16_t >  >::registerInSubclassFactory();
	kernel < 4,         int32_t,       unary_models::constant< 4, int32_t >  >::registerInSubclassFactory();
}

#endif /* __HIPCC__ */

} // namespace model
} // namespace decompression
} // namespace kernels
} // namespace cuda
