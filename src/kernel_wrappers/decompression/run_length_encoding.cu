
#include "kernel_wrappers/common.h"
#ifdef __HIPCC__
#include "kernels/decompression/run_length_encoding.cuh"
#endif

namespace cuda {
namespace kernels {
namespace decompression {
namespace run_length_encoding {

// TODO: This currently ignores the possibility of a sorted variant of the kernel

template<unsigned IndexSize, unsigned UncompressedSize, unsigned RunLengthSize>
class kernel_t : public cuda::registered::kernel_t {
public:
	REGISTERED_KERNEL_WRAPPER_BOILERPLATE_DEFINITIONS(kernel_t);

	using uncompressed_type = util::uint_t<UncompressedSize>;
	using run_length_type = util::uint_t<RunLengthSize>;

	launch_configuration_t resolve_launch_configuration(
		device::properties_t           device_properties,
		device_function::attributes_t  kernel_function_attributes,
		size_t                         uncompressed_length,
		size_t                         position_anchoring_period,
		launch_configuration_limits_t  limits) const
#ifdef __HIPCC__
	{
		launch_config_resolution_params_t<
			IndexSize, UncompressedSize, RunLengthSize
		> params(
			device_properties,
			uncompressed_length, position_anchoring_period);

		return cuda::kernels::resolve_launch_configuration(params, limits);
	}
#else
	;
#endif
};

#ifdef __HIPCC__

template<unsigned IndexSize, unsigned UncompressedSize, unsigned RunLengthSize>
launch_configuration_t kernel_t<IndexSize, UncompressedSize, RunLengthSize>::resolve_launch_configuration(
	device::properties_t           device_properties,
	device_function::attributes_t  kernel_function_attributes,
	arguments_type                 extra_arguments,
	launch_configuration_limits_t  limits) const
{
	auto uncompressed_length       = any_cast<size_t>(extra_arguments.at("uncompressed_length"));
	auto position_anchoring_period = any_cast<size_t>(extra_arguments.at("position_anchoring_period"));

	return resolve_launch_configuration(
		device_properties, kernel_function_attributes,
		uncompressed_length, position_anchoring_period,
		limits);
}

template<unsigned IndexSize, unsigned UncompressedSize, unsigned RunLengthSize>
void kernel_t<IndexSize, UncompressedSize, RunLengthSize>::enqueue_launch(
	stream::id_t                   stream,
	const launch_configuration_t&  launch_config,
	arguments_type                 arguments) const
{
	using index_type        = util::uint_t<IndexSize>;
	using uncompressed_type = util::uint_t<UncompressedSize>;
	using run_length_type   = util::uint_t<RunLengthSize>;

	auto decompressed               = any_cast<uncompressed_type*       >(arguments.at("decompressed"               ));
	auto run_data                   = any_cast<const uncompressed_type* >(arguments.at("run_data"                   ));
	auto run_lengths                = any_cast<const run_length_type*   >(arguments.at("run_lengths"                ));
	auto position_anchors           = any_cast<const index_type*        >(arguments.at("position_anchors"           ));
	auto intra_run_anchor_offsets   = any_cast<const run_length_type*   >(arguments.at("intra_run_anchor_offsets"   ));
	auto position_anchoring_period  = any_cast<util::uint_t<IndexSize>  >(arguments.at("position_anchoring_period"  ));
	auto num_anchors                = any_cast<util::uint_t<IndexSize>  >(arguments.at("num_anchors"                ));
	auto num_element_runs           = any_cast<util::uint_t<IndexSize>  >(arguments.at("num_element_runs"           ));
	auto uncompressed_length        = any_cast<util::uint_t<IndexSize>  >(arguments.at("uncompressed_length"        ));

	cuda::kernel::enqueue_launch(
		*this, stream, launch_config,
		decompressed, run_data, run_lengths, position_anchors, intra_run_anchor_offsets,
		position_anchoring_period, num_anchors, num_element_runs, uncompressed_length
	);
}

template<unsigned IndexSize, unsigned UncompressedSize, unsigned RunLengthSize>
const device_function_t kernel_t<IndexSize, UncompressedSize, RunLengthSize>::get_device_function() const
{
	return reinterpret_cast<const void*>(
		cuda::kernels::decompression::run_length_encoding::decompress<IndexSize, UncompressedSize, RunLengthSize>);
}


static_block {
	//         IndexSize   UncompressedSize  RunLengthSize
	//----------------------------------------------------------------------
	kernel_t < 4,          1,                1 >::registerInSubclassFactory();
	kernel_t < 4,          2,                1 >::registerInSubclassFactory();
	kernel_t < 4,          4,                1 >::registerInSubclassFactory();
	kernel_t < 4,          8,                1 >::registerInSubclassFactory();

	kernel_t < 4,          1,                2 >::registerInSubclassFactory();
	kernel_t < 4,          2,                2 >::registerInSubclassFactory();
	kernel_t < 4,          4,                2 >::registerInSubclassFactory();
	kernel_t < 4,          8,                2 >::registerInSubclassFactory();

	kernel_t < 4,          1,                4 >::registerInSubclassFactory();
	kernel_t < 4,          2,                4 >::registerInSubclassFactory();
	kernel_t < 4,          4,                4 >::registerInSubclassFactory();
	kernel_t < 4,          8,                4 >::registerInSubclassFactory();

	kernel_t < 8,          4,                1 >::registerInSubclassFactory();
	kernel_t < 8,          4,                2 >::registerInSubclassFactory();
	kernel_t < 8,          4,                4 >::registerInSubclassFactory();

	kernel_t < 8,          1,                8 >::registerInSubclassFactory();
	kernel_t < 8,          4,                8 >::registerInSubclassFactory();
	kernel_t < 8,          8,                8 >::registerInSubclassFactory();
}
#endif /* __HIPCC__ */

} // namespace run_length_encoding
} // namespace decompression
} // namespace kernels
} // namespace cuda

