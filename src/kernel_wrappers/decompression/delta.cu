
#include "kernel_wrappers/common.h"
#ifdef __HIPCC__
#include "kernels/decompression/delta.cuh"
#endif

namespace cuda {
namespace kernels {
namespace decompression {
namespace delta {

// TODO: This currently ignores the possibility of a sorted variant of the kernel

template<unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize>
class kernel_t : public cuda::registered::kernel_t {
public:
	REGISTERED_KERNEL_WRAPPER_BOILERPLATE_DEFINITIONS(kernel_t);

	using compressed_type   = uint_t<CompressedSize>;
	using uncompressed_type = uint_t<UncompressedSize>;
	using size_type         = size_type_by_index_size<IndexSize>;

	launch_configuration_t resolve_launch_configuration(
		device::properties_t           device_properties,
		device_function::attributes_t  kernel_function_attributes,
		size_t                         length,
		size_t                         segment_length,
		launch_configuration_limits_t  limits) const
#ifdef __HIPCC__
	{
		launch_config_resolution_params_t<
			IndexSize, UncompressedSize, CompressedSize
		> params(
			device_properties,
			length, segment_length,
			limits.dynamic_shared_memory);

		return cuda::kernels::resolve_launch_configuration(params, limits);
	}
#else
	;
#endif

};

#ifdef __HIPCC__

template<unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize>
inline launch_configuration_t kernel_t<IndexSize, UncompressedSize, CompressedSize>::resolve_launch_configuration(
	device::properties_t           device_properties,
	device_function::attributes_t  kernel_function_attributes,
	arguments_type                 extra_arguments,
	launch_configuration_limits_t  limits) const
{
	auto length         = any_cast<size_t>(extra_arguments.at("length"        ));
	auto segment_length = any_cast<size_t>(extra_arguments.at("segment_length"));

	return resolve_launch_configuration(
		device_properties, kernel_function_attributes,
		length, segment_length,
		limits);
}

template<unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize>
inline void kernel_t<IndexSize, UncompressedSize, CompressedSize>::enqueue_launch(
		stream::id_t                   stream,
		const launch_configuration_t&  launch_config,
		arguments_type                 arguments) const
{
	using compressed_type   = uint_t<CompressedSize>;
	using uncompressed_type = uint_t<UncompressedSize>;

	auto decompressed      = any_cast<uncompressed_type*       >(arguments.at("decompressed"    ));
	auto compressed_input  = any_cast<const compressed_type*   >(arguments.at("compressed_input"));
	auto anchor_values     = any_cast<const uncompressed_type* >(arguments.at("anchor_values"   ));
	auto length            = any_cast<util::uint_t<IndexSize>  >(arguments.at("length"          ));
	auto segment_length    = any_cast<util::uint_t<IndexSize>  >(arguments.at("segment_length"  ));

	cuda::kernel::enqueue_launch(
		*this, stream, launch_config,
		decompressed, compressed_input, anchor_values, length, segment_length
	);
}

template<unsigned IndexSize, unsigned UncompressedSize, unsigned CompressedSize>
inline const device_function_t kernel_t<IndexSize, UncompressedSize, CompressedSize>::get_device_function() const
{
	return cuda::kernels::decompression::delta::decompress<IndexSize, UncompressedSize, CompressedSize>;
}

static_block {
	//         IndexSize   UncompressedSize   CompressedSize
	// ----------------------------------------------------
	kernel_t < 4,          2,                 1 >::registerInSubclassFactory();
	kernel_t < 4,          4,                 1 >::registerInSubclassFactory();
	kernel_t < 4,          4,                 2 >::registerInSubclassFactory();
	kernel_t < 4,          8,                 1 >::registerInSubclassFactory();
	kernel_t < 4,          8,                 2 >::registerInSubclassFactory();
	kernel_t < 4,          8,                 4 >::registerInSubclassFactory();

	kernel_t < 8,          2,                 1 >::registerInSubclassFactory();
	kernel_t < 8,          4,                 1 >::registerInSubclassFactory();
	kernel_t < 8,          4,                 2 >::registerInSubclassFactory();
	kernel_t < 8,          8,                 1 >::registerInSubclassFactory();
	kernel_t < 8,          8,                 2 >::registerInSubclassFactory();
	kernel_t < 8,          8,                 4 >::registerInSubclassFactory();

}

#endif /* __HIPCC__ */

} // namespace delta
} // namespace decompression
} // namespace kernels
} // namespace cuda
